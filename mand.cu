#include "hip/hip_runtime.h"
#include <iostream>
#include "libbmp/CPP/libbmp.h"

#define NUM_ITER 1000
#define GRID_RES 0.001
#define GRID_DIM ((int) (4 / GRID_RES) + 1)

__device__ int solve(float x, float y)
{
    float z_a = 0, z_b = 0;
    for (int i = 0; i < NUM_ITER; i++) {
	if (z_a * z_a + z_b * z_b > 4) {
	    int val = (int) (255 * ((float) i / 16));
	    return (val > 255) ? 255 : val;
	}

	float w_a = (z_a*z_a - z_b*z_b) + x;
	float w_b = 2 * z_a * z_b + y;
	z_a = w_a;
	z_b = w_b;
    }

    return 0;
}

__global__ void mandelbrot(int* grid, int n, float res)
{    
    int origin = n / 2;
    //float res2 = res / 2;
    int stride = blockDim.x;
    for (int row = threadIdx.x; row < n; row += stride) {
	float y = (row - origin) * res;
	for (int col = 0; col < n; col++) {
	    float x = (col - origin) * res;
	    //printf("(x=%f,y=%f)\n", x, y);
	    grid[row*n+col] = solve(x,y);
	}
    }
}

int main()
{
    int* grid = (int*) malloc(GRID_DIM * GRID_DIM * sizeof(int));
    int* grid_dev;
    if (hipMalloc(&grid_dev, GRID_DIM*GRID_DIM*sizeof(int)) != hipSuccess) {
	printf("Could not allocate enough memory\n");
	return 1;
    }
    mandelbrot<<<1, 256>>>(grid_dev, GRID_DIM, GRID_RES);
    hipMemcpy(grid, grid_dev, GRID_DIM*GRID_DIM*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(grid_dev);

    BmpImg img(GRID_DIM, GRID_DIM);
    //bmp_img_init_df(&img, GRID_DIM, GRID_DIM);
    for (int i = 0; i < GRID_DIM; i++) {
	for (int j = 0; j < GRID_DIM; j++) {
	    int val = grid[i*GRID_DIM+j];
	    img.set_pixel(j, i, 0, 0, val);
	    //printf("(%d,%d): %d\n", i, j, (&grid[i])[j]);
//img.set_pixel(j, i, 255, 255, 255);
	}
    }
    //bmp_img_write(&img, "mandelbrot.bmp");
    //bmp_img_free(&img);
    img.write("test.bmp");
    free(grid);
    return 0;
}