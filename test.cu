
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("%d KiB free, %d KiB total\n", free/1024, total/1024);
    return 0;
}